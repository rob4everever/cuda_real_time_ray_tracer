#include "hip/hip_runtime.h"
/*
* author: 100086865
*
* The main file of the ray tracer. GLFW, OpenGL and CUDA are initialised here.
* The ray tracer is configured, executed and the results are rendered to the
* window.
*/

#define GLM_FORCE_CUDA
#define GLM_ENABLE_EXPERIMENTAL

#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <glm/glm.hpp>
#include "cuda_gl_interop.h"

#include <fstream>
#include <iostream>
#include <time.h>

#include "Camera.cuh"
#include "Material.cuh"
#include "Ray.cuh"
#include "Rectangle.cuh"
#include "Scene.cuh"
#include "SceneObject.cuh"
#include "Sphere.cuh"
#include "Texture.cuh"

//Screen dimensions
const unsigned int SCR_WIDTH = 512;
const unsigned int SCR_HEIGHT = 256;

//CUDA/OpenGL interop resources
GLuint viewGLTexture;
hipGraphicsResource_t viewCudaResource;
float* deviceRes = NULL;

//Callbacks
void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode);
GLFWwindow* initGLFW();
void initGL();
void initCuda();
void callCudaKernel(hipSurfaceObject_t image);

//Camera settings
glm::vec3 camLookFrom = glm::vec3(-8, 4, 4);
glm::vec3 camLookAt = glm::vec3(0, 0, -1);

//Current scene
int sceneNumber = 2;

//macro to output hipError_t result to stdout
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
static void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << " " << hipGetErrorString(result) << "' \n";
		hipDeviceReset();
		exit(99);
	}
}

/*
* Determines the colour value of a pixel
* @param: Ray at which to determine the pixel colour
* @param: Scene to fire the ray into
* @param: RandomState
* @return: RGB colour value
*/
__device__ glm::vec3 calculateColour(const Ray& ray, SceneObject** world, hiprandState* local_rand_state) {

	Ray currentRay = ray;
	glm::vec3 currentAttenuation = glm::vec3(1.0f, 1.0f, 1.0f);

	for (int i = 0; i < 50; i++) {

		SceneObjectRecord record;
		//intersect
		if ((*world)->intersect(currentRay, 0.001f, FLT_MAX, record)) {
			Ray scattered;
			glm::vec3 attenuation;
			if (record.material->scatter(currentRay, record, attenuation, scattered, local_rand_state)) {
				currentAttenuation *= attenuation;
				currentRay = scattered;
			}
			else {
				return glm::vec3(0, 0, 0);
			}
		}
		//background colour
		else {
			glm::vec3 unitDir = glm::normalize(currentRay.direction);
			float t = 0.5f * (unitDir.y + 1.0f);
			glm::vec3 x = (1.0f - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
			return currentAttenuation * x;
		}
	}
}

/*
* Initialises the random state for each pixel
* @param: Max x coordinate
* @param: Max y coordinate
* @param: RandomState
*/
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	const int pixelIndex = (j * max_x + i);
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixelIndex, 0, &rand_state[pixelIndex]);
}

/*
* Caclautes an image using the ray tracer and writes the output to a cuda surface
* @param: Cuda surface object to be written to
* @param: Image x
* @param: Image y
* @param: Number of samples to do
* @param: Scene camera
* @param: Scene
* @param: RandomState
* @param: device Res
*/
__global__ void render(hipSurfaceObject_t image, int x, int y, int samples, Camera** camera, SceneObject** scene, hiprandState* rand_state, float* deviceRes) {

	//coordinates of each thread
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= x) || (j >= y)) return;

	//current pixel
	int pixelIndex = j * x + i;
	hiprandState local_rand_state = rand_state[pixelIndex];

	//samples the colour
	glm::vec3 colour = glm::vec3(0.0f, 0.0f, 0.0f);
	for (int s = 0; s < samples; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(y);
		//Cast a ray into the center of the pixel
		Ray ray = (*camera)->getRay(u, v);
		colour += calculateColour(ray, scene, &local_rand_state);
	}

	//average the colour
	rand_state[pixelIndex] = local_rand_state;
	colour.x = colour.x / samples;
	colour.y = colour.y / samples;
	colour.z = colour.z / samples;

	//square rooting the result to solve gamma correction issues
	colour = glm::vec3(sqrt(colour.x), sqrt(colour.y), sqrt(colour.z));

	//write to the surface object
	uchar4 data = make_uchar4(255.99 * colour.x, 255.99 * colour.y, 255.99 * colour.z, 1);
	surf2Dwrite(data, image, i * sizeof(uchar4), j, hipBoundaryModeClamp);
}

/*
* Initialises a scene of objects
* @param: List of scene objects
* @param: World to store the scene
* @param: Scene camera
* @param: Image x
* @param: Image y
*/
__global__ void createWorld(SceneObject** d_list, SceneObject** d_world, Camera** camera, int x, int y, glm::vec3 c1, glm::vec3 c2, int sceneNumber) {

	if (threadIdx.x == 0 && blockIdx.x == 0) {

		if (sceneNumber == 1) {
			Material* light = new Light(new ColouredTexture(glm::vec3(15.0f, 15.0f, 15.0f)));
			Texture* checkedTexture = new  CheckeredTexture(new ColouredTexture(glm::vec3(0, 0, 0)), new ColouredTexture(glm::vec3(1, 1, 1)));
			d_list[0] = new Sphere(glm::vec3(0, 0, -1), 0.5f, new Diffuse(new ColouredTexture(glm::vec3(0.4f, 0.7f, 0.1f))));
			d_list[1] = new Sphere(glm::vec3(0, -1000.5, -1), 1000.0f, new Diffuse(checkedTexture));
			d_list[2] = new Sphere(glm::vec3(1, 0, -1), 0.5f, new Specular(glm::vec3(0.8f, 0.6f, 0.2f)));
			d_list[3] = new Sphere(glm::vec3(-1, 0, -1), 0.5f, new Fresnel(1.5));
			d_list[4] = new Sphere(glm::vec3(-1, 0, -1), -0.45f, new Fresnel(1.5));
			d_list[5] = new XZ_Rectangle(446 * 2, 1046 * 4, 494 * 4, 1024 * 4, 1200 * 4, light);
			*d_world = new Scene(d_list, 6);
			*camera = new Camera(glm::vec3(c1.x, 4, 4), glm::vec3(0, 0, -1), glm::vec3(0, 1, 0), 40.0f, float(x) / float(y));
		}
		if (sceneNumber == 2) {
			int i = 0;
			Material* red = new Diffuse(new ColouredTexture(glm::vec3(0.65f, 0.05f, 0.05f)));
			Material* white = new Diffuse(new ColouredTexture(glm::vec3(0.73f, 0.73f, 0.73f)));
			Material* green = new Diffuse(new ColouredTexture(glm::vec3(0.12f, 0.45f, 0.15f)));
			Material* purple = new Diffuse(new ColouredTexture(glm::vec3(0.52f, 0.25f, 0.85f)));
			Material* orange = new Diffuse(new ColouredTexture(glm::vec3(0.72f, 0.35f, 0.15f)));

			Material* r = new Specular(glm::vec3(0.82f, 0.35f, 0.24f));
			Material* reflectiveblue = new Specular(glm::vec3(0.24f, 0.35f, 0.9f));
			Material* reflectiveYellow = new Specular(glm::vec3(0.99f, 0.98f, 0.1f));

			Material* light = new Light(new ColouredTexture(glm::vec3(15.0f, 15.0f, 15.0f)));

			//cornells box
			d_list[i++] = new FlipNormals(new  YZ_Rectangle(0.0f, 1000.0f, 0.0f, 1000.0f, 1000.0f, green));
			d_list[i++] = new YZ_Rectangle(0.0f, 1000.0f, 0.0f, 1000.0f, 0.0f, red);
			d_list[i++] = new XZ_Rectangle(0.0f, 1000.0f, 0.0f, 1000.0f, 1000.0f, white);						//roof
			d_list[i++] = new XZ_Rectangle(0.0f, 1000.0f, 0.0f, 1000.0f, 0.0f, white);						//floor
			d_list[i++] = new FlipNormals(new XY_Rectangle(0, 1000.0f, 0, 1000.0f, 1000.0f, white));

			d_list[i++] = new Sphere(glm::vec3(180.0f, 120.0f, 378.0f), 120.0f, reflectiveblue);
			d_list[i++] = new Sphere(glm::vec3(400.0f, 120.0f, 198.0f), 120.0f, purple);
			d_list[i++] = new Sphere(glm::vec3(700.0f, 60.0f, 90.0f), 60.0f, orange);
			d_list[i++] = new Sphere(glm::vec3(840.0f, 60.0f, 90.0f), 60.0f, reflectiveYellow);
			d_list[i++] = new Sphere(glm::vec3(720.0f, 120.0f, 621.0f), 120.0f, new Fresnel(1.50f));
			d_list[i++] = new Sphere(glm::vec3(720.0f, 120.0f, 621.0f), 115.0f, new Fresnel(1.50f));
			d_list[i++] = new Sphere(glm::vec3(170.0f, 50.0f, 170.0f), 50.0f, new Fresnel(1.50f));
			d_list[i++] = new Sphere(glm::vec3(170.0f, 50.0f, 170.0f), 45.0f, new Fresnel(1.50f));
			d_list[i++] = new XZ_Rectangle(446 * 2, 1046 * 4, 494 * 4, 1024 * 4, 1200 * 4, light);

			*d_world = new Scene(d_list, i);
			*camera = new Camera(glm::vec3(0.0f + c1.x, 278.0f, -800.0f), glm::vec3(0.0f + c2.x, 278.0, 0.0f), glm::vec3(0, 1, 0), 40.0f, float(x) / float(y));
		}
	}
}

/*
* Deletes the scene of objects
* @param: List of scene objects
* @param: The scene
* @param: Camera
*/
__global__ void clearScene(SceneObject** d_list, SceneObject** d_scene, Camera** d_camera) {

	for (int i = 0; i < 14; i++) {
		delete d_list[i];
	}

	delete* d_scene;
	delete* d_camera;
}

/*
* Initialises the GLFW window
* @return - window object
*/
GLFWwindow* initGLFW() {

	//GLFW configuration
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_COMPAT_PROFILE);

	//GLFW window
	GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "Real Time Ray Tracer", NULL, NULL);
	if (window == NULL) {
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return NULL;
	}

	glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
	glfwSetKeyCallback(window, key_callback);

	return window;
}

/*
* Initialises OpenGL and create a new texture
* to store the rendered image
*/
void initGL() {

	//GLAD
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
		std::cout << "Failed to initialize GLAD" << std::endl;
	}

	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &viewGLTexture);
	glBindTexture(GL_TEXTURE_2D, viewGLTexture);
	{
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, SCR_WIDTH, SCR_HEIGHT, 0, GL_RGB, GL_FLOAT, NULL);
		glBindTexture(GL_TEXTURE_2D, 0);
	}
}

/*
* Initialises cuda and allocates memory on the GPU
* for the writable surface object
*/
void initCuda() {
	checkCudaErrors(cudaGLSetGLDevice(0));
	checkCudaErrors(hipGraphicsGLRegisterImage(&viewCudaResource, viewGLTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
	checkCudaErrors(hipMalloc((void**)&deviceRes, SCR_WIDTH * SCR_HEIGHT * 4 * sizeof(float)));
}

/*
* Sets up the ray tracer and calls the CUDA kernel that
* begins the ray tracing
* @param: surface object to be writtent to
*/
void callCudaKernel(hipSurfaceObject_t image) {

	//number of samples per pixel
	int samples = 50;

	//allocate random state
	hiprandState* d_rand_state;
	checkCudaErrors(hipMalloc((void**)&d_rand_state, SCR_WIDTH * SCR_HEIGHT * sizeof(hiprandState)));

	//define number of threads per block
	int xThreads = 8;
	int yThreads = 8;

	//List of objects
	SceneObject** d_list;
	checkCudaErrors(hipMallocManaged((void**)&d_list, 14 * sizeof(SceneObject*)));

	//Complete scene
	SceneObject** d_scene;

	//Camera
	Camera** d_camera;
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));
	checkCudaErrors(hipMallocManaged((void**)&d_scene, sizeof(SceneObject*)));

	//different camera movements for different scenes
	if (sceneNumber == 1) {
		if (camLookFrom.x <= 20.0f) {
			camLookFrom += 0.2f;
		}
		else {
			camLookFrom.x = -20.0f;
		}
	}
	if (sceneNumber == 2) {

		if (camLookFrom.x >= 1000.0f) {
			camLookFrom.x = 0.0f;
			camLookAt.x = 0.0f;
		}
		else {
			camLookAt += 5.0f;
			camLookFrom += 5.0f;
		}
	}

	//initialise the world
	createWorld << <1, 1 >> > (d_list, d_scene, d_camera, SCR_WIDTH, SCR_HEIGHT, camLookFrom, camLookAt, sceneNumber);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	//initialise blocks and threads
	dim3 blocks((SCR_WIDTH - 1) / xThreads + 1, (SCR_HEIGHT - 1) / yThreads + 1, 1);
	dim3 threads(xThreads, yThreads, 1);

	//init render
	render_init << <blocks, threads >> > (SCR_WIDTH, SCR_HEIGHT, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();

	//threads launch in blocks. block has 64 threads running the function
	render << <blocks, threads >> > (image, SCR_WIDTH, SCR_HEIGHT, samples, d_camera, d_scene, d_rand_state, deviceRes);				//start ray tracing
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";

	//delete scene objects
	clearScene << <1, 1 >> > (d_list, d_scene, d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_scene));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_rand_state));
}

/*
* Deletes cuda resources
*/
void deInit() {
	checkCudaErrors(hipFree(deviceRes));
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipDeviceReset());
}

/*
* Initialises OpenGL, CUDA and GLFW as well as holds
* the main render loop for the application
*/
int main() {

	//setup 
	GLFWwindow* window = initGLFW();
	initGL();
	initCuda();

	//choose a scene
	std::cout << "Please choose a scene: " << std::endl;
	std::cout << "1) 3 spheres" << std::endl;
	std::cout << "2) Cornell's Box" << std::endl;
	std::cin >> sceneNumber;

	//Game loop
	while (!glfwWindowShouldClose(window)) {

		//poll events
		glfwPollEvents();

		//clear colour
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glColor3f(1.0f, 1.0f, 1.0f);

		//map cuda resource
		checkCudaErrors(hipGraphicsMapResources(1, &viewCudaResource));

		hipArray_t viewCudaArray;
		checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, viewCudaResource, 0, 0));

		hipResourceDesc viewCudaArrayResourceDesc;
		memset(&viewCudaArrayResourceDesc, 0, sizeof(viewCudaArrayResourceDesc));
		viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
		viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

		//create a writable cuda surface
		hipSurfaceObject_t viewCudaSurfaceObject;
		checkCudaErrors(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));

		//init the ray tracer and call the kernel
		callCudaKernel(viewCudaSurfaceObject);

		//clean up
		checkCudaErrors(hipDestroySurfaceObject(viewCudaSurfaceObject));
		checkCudaErrors(hipGraphicsUnmapResources(1, &viewCudaResource));
		checkCudaErrors(hipStreamSynchronize(0));

		//bind texture
		glEnable(GL_TEXTURE_2D);
		glBindTexture(GL_TEXTURE_2D, viewGLTexture);

		//draw textured quad
		glBegin(GL_QUADS);
		glTexCoord2f(0.0f, 1.0f); glVertex2f(+1.0f, +1.0f);
		glTexCoord2f(1.0f, 1.0f); glVertex2f(-1.0f, +1.0f);
		glTexCoord2f(1.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
		glTexCoord2f(0.0f, 0.0f); glVertex2f(+1.0f, -1.0f);
		glEnd();

		//unbind texture
		glBindTexture(GL_TEXTURE_2D, 0);
		glFinish();

		//Swap buffers
		glfwSwapBuffers(window);
	}

	//deallocate resources
	deInit();
	glfwTerminate();
	return 0;
}

/*
* Processes keyboard input
*/
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode) {

	//ESC: closes the application
	if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
		glfwSetWindowShouldClose(window, GL_TRUE);
	}
}

/*
* Resize viewport
*/
void framebuffer_size_callback(GLFWwindow* window, int width, int height) {
	glViewport(0, 0, width, height);
}